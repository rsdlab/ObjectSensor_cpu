
#include <iostream>
#include "cutil.h"
#include "cuda_image.h"

// ===================================================================
// ===================================================================

void DevImage::clear(void)
{
  if (this->data) hipFree( this->data );
  this->data = NULL;
  this->w = this->h = 0;
}

bool DevImage::setImage(int w, int h)
{
  if (w <= 0 || h <= 0) { clear(); return true; }
  if (this->w != w || this->h != h || this->data == NULL) {
    clear();
    int total = w * h * sizeof(float);
    hipMalloc((void**)&this->data, total);
    if (this->data == NULL) return false;
    this->w = w;  this->h = h;
  }
  return true;
}

bool DevImage::downloadFrom(float *src_buffer)
{
  if (!src_buffer || !this->data) return false;
  int total = this->w * this->h * sizeof(float);
  hipMemcpy( this->data, src_buffer, total, hipMemcpyHostToDevice );
  return true;
}

bool DevImage::uploadTo(float *dst_buffer)
{
  if (!dst_buffer || !this->data) return false;
  int total = this->w * this->h * sizeof(float);
  hipMemcpy( dst_buffer, this->data, total, hipMemcpyDeviceToHost );
  return true;
}

void DevImage::printInfo(char *cmmt)
{
  printf("%s (%4d x %4d) Device Image\n", (cmmt ? cmmt : "CUDAIMG"), this->w, this->h);
}

// ===================================================================
// ===================================================================

void HstImage::clear(void)
{
  if (this->data) free( this->data );
  this->data = NULL;
  this->w = this->h = this->hmax = 0;
}

bool HstImage::setImage(int w, int h)
{
  if (w <= 0) { clear(); return true; }
  int tn = w * h, to = this->w * this->hmax;
  if (tn > to) {
    if (!reallocImage( w, h )) return false;
  }
  this->w = w;  this->h = h;
  return true;
}

bool HstImage::reallocImage(int w, int hmax)
{
  if (w <= 0 || hmax <= 0) { clear(); return true; }
  int total = w * hmax * sizeof(float);
  this->data = (float*)realloc( this->data, total );
  if (this->data) { 
    this->w = w;  this->hmax = hmax;
    return true;
  } else {
    this->w = this->h = this->hmax = 0;
    return false;
  }
}

bool HstImage::clearImage(float v)
{
  int  i, total = w * h;
  if (data) for (i=0; i<total; i++) data[i] = v;
  return true;
}

bool HstImage::downloadTo(DevImage *cimg)
{
  if (!this->data || !cimg || !cimg->data) return false;
  int total = this->w * this->h * sizeof(float);
  hipMemcpy( cimg->data, this->data, total, hipMemcpyHostToDevice );
  return true;
}

bool HstImage::uploadFrom(DevImage *cimg)
{
  if (!this->data || !cimg || !cimg->data) return false;
  int total = this->w * this->h * sizeof(float);
  hipMemcpy( this->data, cimg->data, total, hipMemcpyDeviceToHost );
  return true;
}

float* HstImage::yieldData(void)
{
  float *dbuffer = this->data;
  this->data = NULL;
  this->w = this->h = this->hmax = 0;
  return dbuffer;
}

void HstImage::printInfo(char *cmmt, char *fmt, int px, int py, int pw, int ph)
{
  int k, nsp=0, len=(cmmt!=NULL?strlen(cmmt):0);  char spaces[41];
  if (cmmt!=NULL) { for (k=0; k<40&&k<len; k++) if (cmmt[k]==' ') nsp++; }
  for (k=0; k<nsp; k++) spaces[k] = ' ';  spaces[k] = '\0';
  if (pw * ph <= 0) {
    printf("%s (%4d x %4d / %d)\n", (cmmt!=NULL ? cmmt : "HOSTImage"), w, h, hmax);
  } else {
    printf("%s (%4d x %4d / %d) (%d,%d:%dx%d)\n", (cmmt ? cmmt : "HOSTImage"), w, h, hmax, px, py, pw, ph);
    if (fmt  ==NULL) fmt = (char*)"%12g ";
    for (int j=0; j < this->h; j++) {
      if (j < py) continue; if (j >= py+ph) break;
      printf("%s  ", spaces);
      for (int i=0; i < this->w; i++)
	if (i >= px && i < px+pw) printf(fmt, this->data[j*this->w+i]);
      printf("\n");
    }
  }
}

// ===================================================================
// ===================================================================

void DevTexture::clear(void)
{
  if (this->array) hipFreeArray( (hipArray*)this->array ); 
  this->array = NULL;
  this->w = this->h = 0;
}

bool DevTexture::setTexture(int w, int h)
{
  hipChannelFormatDesc t_desc = hipCreateChannelDesc<float>(); 
  hipMallocArray((hipArray **)&this->array, &t_desc, w, h);
  if (this->array == NULL) { this->w = this->h = 0;  return false; }
  this->w = w;  this->h = h;
  return true;
}

bool DevTexture::copyFrom(DevImage *cimg)
{
  if (!this->array || !cimg || !cimg->data) return false;
  hipMemcpyToArray((hipArray*)this->array, 0, 0, 
		    cimg->data, sizeof(float)*cimg->w*cimg->h, 
		    hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
  return true;
}

bool DevTexture::copyFrom(HstImage *himg)
{
  if (!this->array || !himg || !himg->data) return false;
  hipMemcpyToArray((hipArray*)this->array, 0, 0, 
		    himg->data, sizeof(float)*himg->w*himg->h, 
		    hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  return true;
}

// ===================================================================
// ===================================================================

bool cuda_init(void)
{
  static bool first_time = true;
  if (first_time) {
//     char  *argv[]={"--quiet"};
//     CUT_DEVICE_INIT( 1, argv );  //// initialize the data structures
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
        return false;
    }
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (deviceProp.major < 1) {
        fprintf(stderr, "cutil error: device does not support CUDA.\n");
        return false;
    }
    hipSetDevice(0);
  }
  first_time = false;
  return true;
}

void timer_start(unsigned int *hTimer)
{
  cutCreateTimer(hTimer);
  cutResetTimer(*hTimer);
  cutStartTimer(*hTimer);
}

double timer_stop(unsigned int hTimer)
{
  cutStopTimer(hTimer);
  double gpuTime = cutGetTimerValue(hTimer);
  cutDeleteTimer(hTimer);
  return gpuTime;
}

