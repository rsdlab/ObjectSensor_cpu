#include "hip/hip_runtime.h"

#include <stdio.h>
#include "cutil.h"
#include "../common/vm_macros.h"
#include "cuda_image_edit.h"
#include "cuda_image_filter.h"
#include "cuda_sift.h"

#define Pi 3.1415926536f
__device__ __constant__ float sift_const[4];
static bool cds_verbose = false;

void check_img(DevImage *cimg, char *cmmt, int oct, int lvl, bool print_on, bool save_on);
void check_sift(DevImage *sift, int cnt, int *ptrs=NULL, int iw=0, int ih=0);
int  csift_extract_recur(HstImage *flist, DevImage *img, int nOctaves, int nLevels,
			 float sigma, float ethreshold, float subsampling, int octave, int xywh[4]);

// ===================================================================
// ===================================================================

bool cuda_sift_extract(DevImage *img, HstImage *flist, int nOctaves, int nLevels,
		       float sigma, float ethreshold, int xywh[4], bool verbose)
{
  // Extract SIFT features from input image 'img', and save them in 'flist'.
  //   'img'  : FLOAT image on CUDA device memory with arbitrary width and height.
  //   'flist': FLOAT image on host memory with size 140 x N, 
  //            each row representing a SIFT feature, totaling N features.
  //            ( x,y,s,sh,  edge,ori,sco,amb,  m,mx,my,me,  desc[128] )
  if (!img || img->w <= 0 || img->h <= 0 || img->data == NULL) {
    printf("SIFT ERROR: invalid input image\n");
    return false;
  }
  int    w = img->w,  h = img->h;
  cds_verbose = verbose;
  // initialize the data structure for SIFT features
  // Note that memory buffer of 'flist' is preserved.
  flist->setImage( 140, 0 );
  
  // decide the number of octaves
  bool upsampling = (nOctaves < 0 ? true : false);
  if (nOctaves < 1) {
    // optionally autoselect the number of octaves. (downsample up to 8x8 patches)
    nOctaves = int(std::floor(log((double)(w<h ? w:h))/log(2.0)) + (upsampling?1:0) -3);
    if (nOctaves < 1) nOctaves = 1;
  }
  // set the original image
  DevImage cimg;
  // set the input image
  int   omin,  xywh2[4];
  float subsampling = 1.0f;
  if (upsampling) {		// input image : up-sampling
    if (cds_verbose) printf("SIFT options: nOct=%d nLev=%d sigma=%.2f eth=%.2f with upsampling\n", nOctaves, nLevels, sigma, ethreshold);
    DevImage timg, bimg;
    cimg.setImage( w*2, h*2 );
    bimg.setImage( w*2, h*2 );
    timg.setImage( w*2, h*2 );
    if (timg.data==NULL) {
      printf("Error (cuda_sift_extract): failed to allocate CUDA device memory\n");
      return false;
    }
    double tt = cimg_scale_up( &bimg, img );
    double ret = cimg_lowpass<7>(&cimg, &bimg, &timg, 1.23*1.23) ;
    omin = -1;  subsampling = 0.5f;
    if (xywh) { 
      xywh2[0]=2*xywh[0]; xywh2[1]=2*xywh[1]; xywh2[2]=2*xywh[2]; xywh2[3]=2*xywh[3];
      xywh = xywh2;
    }
  } else if (sigma > 0) {	// input image : smoothing
    if (cds_verbose) printf("SIFT options: nOct=%d nLev=%d sigma=%.2f eth=%.2f with smoothing\n", nOctaves, nLevels, sigma, ethreshold);
    DevImage timg;
    cimg.setImage( w, h );
    timg.setImage( w, h );
    if (cimg.data==NULL || timg.data==NULL) {
      printf("Error (cuda_sift_extract): failed to allocate CUDA device memory\n");
      return false;
    }
    double ret = cimg_lowpass<7>(&cimg, img, &timg, sigma*sigma) ;
    omin = 0;  subsampling = 1.0f;
    ////CUT_CHECK_ERROR2("cuda_sift_extract 0");
  } else {			// input image : copying
    if (cds_verbose) printf("SIFT options: nOct=%d nLev=%d sigma=%.2f eth=%.2f with copying\n", nOctaves, nLevels, sigma, ethreshold);
    cimg.setImage( w, h );
    if (cimg.data==NULL) {
      printf("Error (cuda_sift_extract): failed to allocate CUDA device memory\n");
      return false;
    }
    cimg_copy( &cimg, img );
  }
  if (cimg.data == NULL) return false;
  // extract the SIFT features
  ////printf("nO=%d nL=%d sig=%g ethresh=%g\n", nOctaves, nLevels, sigma, ethreshold);
  int cnt = csift_extract_recur( flist, &cimg, nOctaves, nLevels,
				 sigma, ethreshold, subsampling, omin, xywh);
  //   HstImage 'flist' is 140 x N float image, with a SIFT feature stored in each row.
  //   Each SIFT feature consists of 140 float values, including x,y,s,o and 128 descriptor.
  //   ( SIFTFeature[140]:  x,y,s,sh,  edge,ori,sco,amb,  desc[128],  dummy[4] )
  if (cds_verbose) { printf("Total number of SIFT features : %d \n", cnt); }
  
  return true;
}


// ===================================================================
// ===================================================================

int csift_extract_octave(HstImage *flist, DevImage *img, int nOctave, int nLevels, 
			 float sigma, float ethreshold, float subsampling, int octave, int xywh[4]);
double csift_find_3d_min_max(DevImage *minmax, DevImage *data1, DevImage *data2, 
			     DevImage *data3, float thresh, int maxPts);
double csift_unpack_pointers(DevImage *minmax, int maxPts, int *ptrs, int *numPts, int xywh[4]);
double csift_compute_positions(DevImage *data1, DevImage *data2, DevImage *data3,
			       int *h_ptrs, DevImage *sift, int numPts, int maxPts,
			       float sigma0, int oct, int lvl, int nLevels);
double csift_remove_edge_points(DevImage *sift, int *initNumPts, int maxPts, float ethreshold, int w, int h, float subsampling);
double csift_compute_orientations(DevImage *img, DevImage *sift, int numPts, int maxPts, float subsampling);
double csift_second_orientations(DevImage *sift, int *initNumPts, int maxPts);
double csift_extract_descriptors(DevTexture *timg, DevImage *sift,
				 DevImage *desc, int numPts, int maxPts, float subsampling);
double csift_add_sift_data(HstImage *flist, float *d_sift, float *d_desc,
			   int numPts, int maxPts, float subsampling);


int csift_extract_recur(HstImage *flist, DevImage *img, int nOctaves, int nLevels,
			float sigma, float ethreshold, float subsampling, int octave, int xywh[4])
{
  	if (octave >= nOctaves) return 0;
	int w=img->w, h=img->h, cnt=0;
  	// extract SIFT feature
  	cnt = csift_extract_octave( flist, img, nOctaves, nLevels,
			      sigma, ethreshold, subsampling, octave, xywh );
  	// call itself recursively, to process another octave
  	if (cds_verbose) { printf("octave:%d, nO:%d\n",octave,nOctaves); }
  	if (octave < nOctaves-1) {
		DevImage subImg;
    	bool ret = subImg.setImage( w/2, h/2 );
    	if (ret) {
      		cimg_scale_down(&subImg, img, 0.2f);
      		if (xywh) {
				int xywh2[4] = { xywh[0]/2, xywh[1]/2, xywh[2]/2, xywh[3]/2 };
				cnt += csift_extract_recur( flist, &subImg, nOctaves, nLevels,
					    sigma, ethreshold, subsampling*2.0f, octave+1, xywh2 );
  			} else {
				cnt += csift_extract_recur( flist, &subImg, nOctaves, nLevels,
				    	sigma, ethreshold, subsampling*2.0f, octave+1, NULL );
      		}
    	} else {
      		CUT_CHECK_ERROR2("csift_extract_recur 0");
      		printf("CUDA Error(csift_extract_recur): failed to allocate device memory oct=%d (%dx%d)\n", octave+1, img->w/2, img->h/2);
    	}
  	}
 	return cnt;
}

int csift_extract_octave(HstImage *flist, DevImage *img, int nOctave, int nLevels, 
			 float sigma, float ethreshold, float subsampling, int octave, int xywh[4])
{
	const int maxPts = 1024*3;
  	int w = img->w; 
  	int h = img->h;
  	DevImage blurImg[3], *bimg1, *bimg0;
  	DevImage diffImg[3];
  	blurImg[0].setImage(w,h);  blurImg[1].setImage(w,h);  blurImg[2].setImage(w,h);
  	diffImg[0].setImage(w,h);  diffImg[1].setImage(w,h);  diffImg[2].setImage(w,h);
  	DevImage tmpImg(w,h);
  	DevTexture textImg(w,h);
  	DevImage minmax(w, iDivUp(h,32));
  	DevImage sift( maxPts, 7 ); // { xpos, ypos, scale, value, edge, orient1, orient2 };
  	DevImage desc( 128, maxPts );
  	if (desc.data == NULL) {
    	printf("CUDA Error(csift_extract_octave): failed to allocate device memory oct=%d (%dx%d)\n", octave, img->w, img->h);
    	return 0;
  	}
  	// calculate the variance of the Gaussian filter for each level
  	float sii[10], sig[10]={0,0,0,0,0, 0,0,0,0,0};
  	if (true) {
    	float sk = powf( 2.0f, 1.0f/nLevels );
    	float sq_k2_1 = sqrtf( sk*sk - 1 );
    	for (int i=0; i<nLevels+3; i++) {
      		sig[i] = sigma * powf( sk, i );		// sigma at each level
      		sii[i] = (i==0 ? 0 : sq_k2_1 * sig[i-1]);	// sigma for incremental filtering
    	}
  	}
  	CUT_CHECK_ERROR2("csift_extract_octave 0");
  	CUDA_SAFE_CALL(hipDeviceSynchronize());
  	float  threshold = 0.04f / nLevels / 2.0f;
  	double etime[]={0,0,0,0,0, 0,0,0,0,0, 0,0};
  	int totPts = 0;
  	int *ptrs = (int *)malloc(sizeof(int)*maxPts);
  	hipError_t err;
  	
  	for (int i=0; i<nLevels+3; i++) {
    	if(i == 0) {
	      	if (cds_verbose) { printf("O=%d/%d L=%d/%d ", octave, nOctave, i, nLevels); check_img(img, "zcBlur", octave, i, true, false); }
    	} else if (i >  0) { 
      		bimg1 = (i==1 ? img : &blurImg[(i-1)%3]);
      		bimg0 = &blurImg[i%3];
      		// blur the next level image
      		etime[0] += cimg_lowpass<7>( bimg0, bimg1, &tmpImg, sii[i]*sii[i] );
			if (cds_verbose) { printf("      L=%d/%d s=%.4f ", i, nLevels, sii[i]); check_img(bimg0, "zcBlur", octave, i, false, false); }
      		// get the difference image
      		etime[1] += cimg_subtract(&diffImg[(i)%3], bimg0, bimg1 );
      		if (cds_verbose) {
      			printf("Diff(%d) ", i%3);
      		}
      		//check_img( &diffImg[(i)%3], "zcDiff", octave, i, false, false );
      		err = hipGetLastError();
      		if (err != hipSuccess) {
      			if (cds_verbose) printf("=> error\n");
  			} else if (i < 3) {
  				if (cds_verbose) printf("\n");
			} else {
				// find the local maxima
				if (cds_verbose) { printf("SIFT(%d,%d,%d) ", (i)%3, (i-1)%3, (i-2)%3); }
				etime[2] += csift_find_3d_min_max(&minmax, &diffImg[(i)%3], &diffImg[(i-1)%3], &diffImg[(i-2)%3], threshold, maxPts);
				int numPts = 0;
				etime[3] += csift_unpack_pointers(&minmax, maxPts, ptrs, &numPts, xywh);
				if (numPts<=0) {
					if (cds_verbose) printf("\n");
				} else {
	  				etime[5] += csift_compute_positions(&diffImg[(i)%3], &diffImg[(i-1)%3], &diffImg[(i-2)%3], 
					      ptrs, &sift, numPts, maxPts, sig[0], octave, i-2, nLevels);
	  				CUT_CHECK_ERROR2("csift_extract_octave 5");
	  				//check_sift( &sift, numPts, ptrs, bimg1->w, bimg1->h );
	  				etime[4] += textImg.copyFrom( &blurImg[(i-2)%3] );
	  				etime[6] += csift_remove_edge_points(&sift, &numPts, maxPts, ethreshold, bimg0->w, bimg0->h, subsampling);
	  				if (cds_verbose) printf("  numPts = %d / %d   scale (%d,%d/%d)\n", numPts, maxPts, octave, (i-2), nLevels);
	  				if (numPts <= 0) {
	  					if (cds_verbose) printf("\n"); continue;
  					}
	  				etime[7] += csift_compute_orientations(&blurImg[(i-2)%3], &sift, numPts, maxPts, subsampling);
	  				etime[8] += csift_second_orientations(&sift, &numPts, maxPts);
	  				etime[9] += csift_extract_descriptors(&textImg, &sift, &desc, numPts, maxPts, subsampling);
	  				etime[10] += csift_add_sift_data(flist, sift.data, desc.data, numPts, maxPts, subsampling);
	  				err = hipGetLastError();
	  				if (err != hipSuccess) numPts = 0;  // CUDA error
	  				if (cds_verbose) {
	    				if (err != hipSuccess) printf("=> error\n");  // CUDA error
	    				else printf("=> %d features\n", numPts);
	  				}
	  				totPts += numPts;
				}
      		}
    	}
    	// copy the blurred image at the top-2 level to the input image,
    	//   so that it could be used as the starting point for the next octave.
    	if (i==nLevels) cimg_copy( img, bimg0 );
  	}
	if (cds_verbose) {
	    printf("  Number of features at this octave: %d\n", totPts);
    	//flist->printInfo("  flist", "%.2f ", 0, flist->h-totPts, 2, totPts);
    	for (int i=0; i<11; i++) etime[11] += etime[i];
    	printf("   Filt  Subt  MiMx  Unpk  Text  Posi  Edge  Orin  Ori2  Desc  Addf   Total \n");
    	printf("  %5.2f %5.2f %5.2f %5.2f %5.2f %5.2f %5.2f %5.2f %5.2f %5.2f %5.2f  %6.2f \n",
	   	etime[0], etime[1], etime[2], etime[3], etime[4], etime[5], etime[6], etime[7], etime[8], etime[9], etime[10], etime[11]);
	}
  	CUT_CHECK_ERROR2("csift_extract_octave 9");
  	CUDA_SAFE_CALL(hipDeviceSynchronize());
  	free(ptrs);
  	
  	return totPts;
}

// ===================================================================
// ===================================================================
#define MINMAX_SIZE   128

__global__ void th_find_3d_min_max(int *d_Result, float *d_Data1, float *d_Data2, 
				   float *d_Data3, int width, int height)
{
  //Data cache
  __shared__ float data1[3*(MINMAX_SIZE + 2)];
  __shared__ float data2[3*(MINMAX_SIZE + 2)];
  __shared__ float data3[3*(MINMAX_SIZE + 2)];
  __shared__ float ymin1[(MINMAX_SIZE + 2)];
  __shared__ float ymin2[(MINMAX_SIZE + 2)];
  __shared__ float ymin3[(MINMAX_SIZE + 2)];
  __shared__ float ymax1[(MINMAX_SIZE + 2)];
  __shared__ float ymax2[(MINMAX_SIZE + 2)];
  __shared__ float ymax3[(MINMAX_SIZE + 2)];

  //Current tile and apron limits, relative to row start
  const int tx = threadIdx.x;
  const int xStart = __mul24(blockIdx.x, MINMAX_SIZE);
  const int xEnd = xStart + MINMAX_SIZE - 1;
  const int xReadPos = xStart + tx - WARP_SIZE;
  const int xWritePos = xStart + tx;
  const int xEndClamped = min(xEnd, width - 1);
  int memWid = MINMAX_SIZE + 2;

  int memPos0 = (tx - WARP_SIZE + 1);
  int memPos1 = (tx - WARP_SIZE + 1);
  int yq = 0;
  unsigned int output = 0;
  for (int y=0;y<34;y++) {

    output >>= 1;
    int memPos =  yq*memWid + (tx - WARP_SIZE + 1);
    int yp = 32*blockIdx.y + y - 1;
    yp = max(yp, 0);
    yp = min(yp, height-1);
    int readStart = __mul24(yp, width);

    //Set the entire data cache contents
    if (tx>=(WARP_SIZE-1)) {
      if (xReadPos<0) {
	data1[memPos] = 0;
	data2[memPos] = 0;
	data3[memPos] = 0;
      } else if (xReadPos>=width) {
	data1[memPos] = 0;
	data2[memPos] = 0;
	data3[memPos] = 0;
      } else {
	data1[memPos] = d_Data1[readStart + xReadPos];
	data2[memPos] = d_Data2[readStart + xReadPos];
	data3[memPos] = d_Data3[readStart + xReadPos];
	//if ((readStart + xReadPos)<0 || (readStart + xReadPos)>=width*height)
	//  printf("csift_find_3d_min_max: read error\n");
      }
    }
    __syncthreads();
  
    int memPos2 = yq*memWid + tx;
    if (y>1) {
      if (tx<memWid) {
	float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
	float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
	float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
	float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
	float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
	float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);
	ymin1[tx] = min1;
	ymin2[tx] = fminf(fminf(min1, min2), min3);
	ymin3[tx] = min3;
	ymax1[tx] = max1;
	ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
	ymax3[tx] = max3;
      }
    }
    __syncthreads();

    if (y>1) {
      if (tx<MINMAX_SIZE) {
	if (xWritePos<=xEndClamped) {
	  float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]), 
	    ymin1[tx+1]), ymin3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
	  minv = fminf(minv, sift_const[1]);
	  float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]),
	    ymax1[tx+1]), ymax3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
	  maxv = fmaxf(maxv, sift_const[0]);
	  if (data2[memPos1+1]<minv || data2[memPos1+1]>maxv)
	    output |= 0x80000000;
	}
      }
    }
    __syncthreads();

    memPos0 = memPos1;
    memPos1 = memPos2;
    yq = (yq<2 ? yq+1 : 0);
  }
  if (tx<MINMAX_SIZE && xWritePos<width) {
    int writeStart = __mul24(blockIdx.y, width) + xWritePos;
    d_Result[writeStart] = output;
    //if (writeStart<0 || writeStart>=width*iDivUp(height,32))
    //  printf("csift_find_3d_min_max: write error\n");
  }
}

double csift_find_3d_min_max(DevImage *minmax, DevImage *data1, DevImage *data2, 
			     DevImage *data3, float thresh, int maxPts)
{
  int *d_res = (int *)minmax->data;
  if (data1->data==NULL || data2->data==NULL || data3->data==NULL || d_res==NULL) {
    printf("Find3DMinMax: missing data %08x %08x %08x %08x\n", 
	   data1->data, data2->data, data3->data, d_res);
    return 0.0;
  }
  int w = data1->w;
  int h = data1->h;
  unsigned int hTimer;
  timer_start(&hTimer);
  float threshs[2] = { +thresh*0.8f, -thresh*0.8f };   ////
  //float threshs[2] = { +0.03f, -0.03f }; ////
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sift_const), &threshs, 2*sizeof(float)));

  dim3 blocks(iDivUp(w, MINMAX_SIZE), iDivUp(h,32));
  dim3 threads(WARP_SIZE + MINMAX_SIZE + 1);
  th_find_3d_min_max<<<blocks, threads>>>(d_res, data1->data, data2->data, data3->data, w, h); 
  CUT_CHECK_ERROR("Find3DMinMax() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("Find3DMinMax time =           %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// ===================================================================

// __global__ void th_unpack_pointers(int *minmax, int *ptrs, int w, int h, int maxPts)
// {
//   const int tx = threadIdx.x;
//   int numPts = 0;
//   for (int y=0;y<h/32;y++) {
//     for (int x=0;x<w;x+=16) {
//       unsigned int val = minmax[y*w+x+tx];
//       if (val) {
// 	for (int k=0;k<32;k++) {
// 	  if (val&0x1 && numPts<maxPts) {
// 	    ptrs[16*numPts+tx] = (y*32+k)*w + x+tx;
// 	    numPts++;
// 	  }
// 	  val >>= 1;
// 	}
//       }
//     }
//   } 
// }

double csift_unpack_pointers(DevImage *minmax, int maxPts, int *ptrs, int *numPts, int xywh[4])
{
  HstImage himg;
  himg.setImage( minmax->w, minmax->h );
  himg.uploadFrom( minmax );
  unsigned int *minmax_data = (unsigned int *)himg.data;
  if (minmax_data==NULL || ptrs==NULL) {
    printf("UnpackPointers: missing data %08x %08x\n", minmax_data, ptrs);
    return 0.0;
  }
  int w = minmax->w;
  int h = 32*minmax->h;
  unsigned int timer = 0;
  timer_start(&timer);
  int num = 0, xymm[4];
  if (xywh) {
    xymm[0]=xywh[0]; xymm[1]=xywh[1]; xymm[2]=xywh[0]+xywh[2]; xymm[3]=xywh[1]+xywh[3]; 
    if (w < xymm[2]) xymm[2] = w;
    if (h < xymm[3]) xymm[3] = h;
  } else { xymm[0]=0; xymm[1]=0; xymm[2]=w; xymm[3]=h; }
  for (int yy=xymm[1]/32;yy<h/32;yy++) {
    if (yy > xymm[3]/32) break;
    for (int x=xymm[0];x<xymm[2];x++) {
      if (xymm[0]==0) { if (x==0 || x == xymm[2]-1) continue; }  // on boundary
      unsigned int val = minmax_data[yy*w+x];
      if (val) {
	//printf("%d %d %08x\n", x, yy, val);
	for (int k=0;k<32;k++) {
	  if (val&0x1 && num<maxPts) {
	    int y = yy * 32 + k;
	    if (xymm[1]==0) { if (y==0 || y == xymm[3]-1) continue; }  // on boundary
	    if (y >= xymm[1] && y < xymm[3]) ptrs[num++] = y*w + x;
	  }
	  val >>= 1;
	}
      }
    }
  }
  *numPts = num;
  double gpuTime = timer_stop(timer);
#ifdef VERBOSE
  printf("UnpackPointers time =         %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// Compute precise positions in xpos, ypos and scale
// ===================================================================
#define POSBLK_SIZE   32

__global__ void th_compute_positions(float *g_Data1, float *g_Data2, float *g_Data3, 
				     int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
  // Calculate exact position of the keypoint on 'g_Data2' and save it in 'd_Sift', 
  //   by fitting 3D quadratic surface to the values in the neighborhood 
  //   including 'g_Data1' (one level above) and 'g_Data3' (below).
  int i = __mul24(blockIdx.x, POSBLK_SIZE) + threadIdx.x;
  if (i>=numPts) return;
  int p = d_Ptrs[i];
  float val[7];
  val[0] = g_Data2[p];
  val[1] = g_Data2[p-1];
  val[2] = g_Data2[p+1];
  float dx = 0.5f*(val[2] - val[1]);
  float dxx = 2.0f*val[0] - val[1] - val[2];
  val[3] = g_Data2[p-w];
  val[4] = g_Data2[p+w];
  float dy = 0.5f*(val[4] - val[3]); 
  float dyy = 2.0f*val[0] - val[3] - val[4];
  val[5] = g_Data3[p];
  val[6] = g_Data1[p];
  float ds = 0.5f*(val[6] - val[5]); 
  float dss = 2.0f*val[0] - val[5] - val[6];
  //// Note this part be hung up if 'p' is invalid, and may return 'unspecified launch failure'
  float dxy = 0.25f*(g_Data2[p+w+1] + g_Data2[p-w-1] - g_Data2[p-w+1] - g_Data2[p+w-1]);
  float dxs = 0.25f*(g_Data3[p+1] + g_Data1[p-1] - g_Data1[p+1] - g_Data3[p-1]);
  float dys = 0.25f*(g_Data3[p+w] + g_Data1[p-w] - g_Data3[p-w] - g_Data1[p+w]);
  float idxx = dyy*dss - dys*dys;
  float idxy = dys*dxs - dxy*dss;  
  float idxs = dxy*dys - dyy*dxs;
  float idyy = dxx*dss - dxs*dxs;
  float idys = dxy*dxs - dxx*dys;
  float idss = dxx*dyy - dxy*dxy;
  float det = idxx*dxx + idxy*dxy + idxs*dxs;
  float idet = 1.0f / det;
  float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
  float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
  float pds = idet*(idxs*dx + idys*dy + idss*ds);
  if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f){
    pdx = __fdividef(dx, dxx);
    pdy = __fdividef(dy, dyy);
    pds = __fdividef(ds, dss);
  }
  // sift_const[] = { sigma0, oct, lvl, nLevels };
  float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
  float refined_level = sift_const[2] + pds;
  float oct = sift_const[1] + refined_level / sift_const[3];	// refined octave
  d_Sift[i+0*maxPts] = (p%w)+0.5f + pdx;			// xpos (on subsampled image)
  d_Sift[i+1*maxPts] = (p/w)+0.5f + pdy;			// ypos (on subsampled image)
  d_Sift[i+2*maxPts] = sift_const[0] * exp2f(oct);		// scale (in original scale)
  d_Sift[i+3*maxPts] = val[0] + dval;				// value
  d_Sift[i+5*maxPts] = 0;					// orientation
  float tra = dxx + dyy;
  det = dxx*dyy - dxy*dxy;
  d_Sift[i+4*maxPts] = __fdividef(tra*tra, det);		// edge strength
  // check if the refined position is valid
  if (pdx > +1.5f || pdx < -1.5f || pdy > +1.5f || pdy < -1.5f) d_Sift[i+2*maxPts] = -99;
  // check if the refined level is valid
  if ( refined_level < sift_const[2]-0.5f || 
       refined_level > sift_const[2]+1.5f ) d_Sift[i+2*maxPts] = -99;
}

double csift_compute_positions(DevImage *diff1, DevImage *diff2, DevImage *diff3,
			       int *h_ptrs, DevImage *sift, int numPts, int maxPts,
			       float sigma0, int oct, int lvl, int nLevels)
{
  // Compute exact position of the keypoints, which are specified by 'h_ptrs'
  //   as pixel indices in the image.
  int w = diff1->w;
  int h = diff1->h;
  unsigned int hTimer;
  timer_start(&hTimer);
  float sparam[4] = { sigma0, oct, lvl, nLevels };
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sift_const), &sparam, 4*sizeof(float)));
  // copy keypoints indices from 'h_ptrs' to 'd_ptrs'
  int *d_ptrs = 0;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_ptrs, sizeof(int)*numPts));
  CUDA_SAFE_CALL(hipMemcpy(d_ptrs, h_ptrs, sizeof(int)*numPts, hipMemcpyHostToDevice));
  
  // calculate sub-pixel location of the keypoints, and save them in 'sift'
  dim3 blocks(iDivUp(numPts, POSBLK_SIZE));
  dim3 threads(POSBLK_SIZE);
  th_compute_positions<<<blocks, threads>>>(diff1->data, diff2->data, diff3->data,
					    d_ptrs, sift->data, numPts, maxPts, w, h);
  
  //CUT_CHECK_ERROR2("ComputePositions() 1 failed");
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  //CUT_CHECK_ERROR2("ComputePositions() 2 failed");
  
  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("ComputePositions time =       %.2f msec\n", gpuTime);
#endif
  CUDA_SAFE_CALL(hipFree(d_ptrs));
  return gpuTime;
}

// ===================================================================
// 
// ===================================================================

double csift_remove_edge_points(DevImage *sift, int *initNumPts, int maxPts, float ethreshold, int w, int h, float subsampling)
{
  unsigned int hTimer;
  timer_start(&hTimer);
  int numPts = *initNumPts;
  int bw = sizeof(float)*numPts;
  float *h_sift = (float *)malloc(5*bw);
  CUDA_SAFE_CALL(hipMemcpy2D(h_sift, bw, sift->data, sizeof(float)*maxPts,  
			      bw, 5, hipMemcpyDeviceToHost));
  float edgeLimit = (ethreshold+1)*(ethreshold+1)/ethreshold; 
  int num = 0;
  for (int i=0;i<numPts;i++) {
    float xx = h_sift[0*numPts+i];                   // x on subsampled image
    float yy = h_sift[1*numPts+i];                   // y on subsampled image
    float scale = h_sift[2*numPts+i] / subsampling;  // scale on subsampled image
    float edgev = h_sift[4*numPts+i];                // edge strength
    //printf("f%d : xx=%.2f yy=%.2f scale=%.2f edgev=%.2f\n", i, xx, yy, scale, edgev);
    if (edgev < edgeLimit && scale > 0 && 
	xx > scale && yy > scale && xx < w-scale && yy < h-scale) {
      for (int j=0;j<5;j++) h_sift[j*numPts+num] = h_sift[j*numPts+i];
      num ++;
    }
  }
  CUDA_SAFE_CALL(hipMemcpy2D(sift->data, sizeof(float)*maxPts, h_sift, bw,  
			      bw, 5, hipMemcpyHostToDevice));
  free(h_sift);
  *initNumPts = num;
  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("RemoveEdgePoints time =       %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// Compute two dominating orientations in xpos and ypos
// ===================================================================
#define OWSZ 16
__global__ void th_compute_orientations(float *g_Data, float *d_finfo, int maxPts, int w, int h)
{
  __shared__ float data[OWSZ*OWSZ];	// OWSZxOWSZ neighbor pixels around the keypoint
  __shared__ float wsum[OWSZ*32];	// 32 x OWSZ weight sum
  const int tx = threadIdx.x;		// OWSZ threads for each keypoint
  const int bx = blockIdx.x;		// bx is just the index of the keypoint
  float xx = d_finfo[0*maxPts + bx];	// refined x position (on subsampled image)
  float yy = d_finfo[1*maxPts + bx];	// refined y position (on subsampled image)
  int yp = ((int)yy) - OWSZ/2;	// Y position of the keypoint - 7
  int xp = ((int)xx) - OWSZ/2;	// X position of the keypoint - 7
  
  // setup shared data (collect pixel values in the 16x16 neighborhood of keypoint (xp+7,yp+7))
  for (int y=0;y<OWSZ;y++) {
    int memPos = OWSZ*y + tx;
    int xi = xp + tx;
    int yi = yp + y;
    if (xi<0) xi = 0;
    if (xi>=w) xi = w-1;
    if (yi<0) yi = 0;
    if (yi>=h) yi = h-1;
    data[memPos] = g_Data[yi*w+xi];
  }
//   { for (int k=0;k<OWSZ;k++) dbuf[k*OWSZ+tx] = 0; }
  __syncthreads();
  
  for (int i=0;i<32;i++) wsum[i*OWSZ+tx] = 0.0f;
  __syncthreads();
  
  {	// calculate the weighted sum of orientations along Y axis
    float sigma = d_finfo[ 2 * maxPts + bx ] / sift_const[0];  // scale (on subsampled image)
    float sigdenom = ( 2 * 1.5f * 1.5f * sigma * sigma );
    //if (bx==0) { dbuf[0] = xp;  dbuf[1] = yp;  dbuf[3] = xx;  dbuf[4] =yy; }
    for (int y=1;y<OWSZ-1;y++) {
      int memPos = y*OWSZ + tx;
      float yoff  = yy - (yp+y+0.5f);  yoff = yoff * yoff;
      if (tx>=1 && tx<OWSZ-1) {
	float dy = data[memPos+OWSZ] - data[memPos-OWSZ];
	float dx = data[memPos+1]  - data[memPos-1];
	float aa = atan2f(dy, dx);   if (aa<0) aa += 2*Pi;
	int bin = (int)(aa*32/(2*Pi)) % (32-1);	// 32 orientations
	float grad = sqrtf(dx*dx + dy*dy);	// gradient magnitude * 2
	// calculate the Grassian falloff
	float xoff = xx - (xp+tx+0.5f);
	float ddist = yoff + xoff * xoff;
	float weight = 0;
	if (ddist <= (OWSZ/2)*(OWSZ/2)) { 
	  weight = exp( -ddist / sigdenom );
	  wsum[bin*OWSZ + tx] += grad * weight;
	}
	//if (bx==0) dbuf[y*OWSZ+tx] = bin;
      }
    }
    //if (bx==0) for (int y=0; y<32; y++) dbuf[y*OWSZ+tx] = wsum[y*OWSZ+tx];
  }
  __syncthreads();
  __shared__ float hist[32*2];
  if (tx < 32/2) {
    // calculate the weighted sum of orientations along X axis
    int k, pos = tx * OWSZ; //__mul24(tx,16);
// #pragma unroll OWSZ-2
    for (k=1; k<=OWSZ-2; k++) wsum[pos] += wsum[ pos + k ];
    hist[tx] = wsum[pos];		// first  half of 32 bins of orientation
    pos = (tx+16) * OWSZ; //__mul24(tx+16,16);
// #pragma unroll OWSZ-2
    for (k=1; k<=OWSZ-2; k++) wsum[pos] += wsum[ pos + k ];
    hist[tx+16] = wsum[pos];	// second half of 32 bins of orientation
    __syncthreads();
    //if (bx==0) { dbuf[tx+0] = hist[tx]; dbuf[tx+16] = hist[tx+16]; }
    {	// smooth the histogram in hist[32 + x]
      float pprv, prev, next, nnxt;
      pprv = hist[ (tx - 2 + 32) & 31 ];	// first half of hist[]
      prev = hist[ (tx - 1 + 32) & 31 ];
      next = hist[ (tx + 1) & 31 ];
      nnxt = hist[ (tx + 2) & 31 ];
      hist[tx + 32] = 6 * hist[tx] + 4 * (prev + next) + (pprv + nnxt);
      pprv = hist[ tx+16 - 2 ];			// second half of hist[]
      prev = hist[ tx+16 - 1 ];
      next = hist[ (tx+16 + 1) & 31 ];
      nnxt = hist[ (tx+16 + 2) & 31 ];
      hist[tx+16 + 32] = 6 * hist[tx+16] + 4 * (prev + next) + (pprv + nnxt);
      __syncthreads();
    }
    {	// find the local maximum in hist[x]
      float v = hist[32+tx];	// first  half of hist[]
      hist[tx] = (v>hist[32+(tx+1)] && v>=hist[32+((tx+31)&31)] ? v : 0.0f);
      v = hist[32+tx+16];	// second half of hist[]
      hist[tx+16] = (v>hist[32+((tx+16+1)&31)] && v>=hist[32+(tx+16-1)] ? v : 0.0f);
    }
  }
  //if (bx==0) { dbuf[tx+0] = hist[tx]; dbuf[tx+16] = hist[tx+16]; }
  __syncthreads();
  // 
  if (tx==0) {
    float *d_Orient = d_finfo + 5*maxPts;  // orientation of the keypoint (degree; 0~360)
    float maxval1 = 0.0;
    float maxval2 = 0.0;
    int i1 = -1;
    int i2 = -1;
    for (int i=0;i<32;i++) {
      float v = hist[i];
      if (v>maxval1) {
	maxval2 = maxval1;
	maxval1 = v;
	i2 = i1;
	i1 = i;
      } else if (v>maxval2) {
	maxval2 = v;
	i2 = i;
      }
    }
    float val1 = hist[32+((i1+1)&31)];
    float val2 = hist[32+((i1+31)&31)];
    float peak = i1 + 0.5f + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
    peak = peak * 0.19634954f;
    if (peak <     0) peak += 2*Pi;
    if (peak >= 2*Pi) peak -= 2*Pi;
    d_Orient[bx] = peak;  // [ 0 ~ 2PI ]
    if (maxval2<0.8f*maxval1)  i2 = -1;
//     if (bx==1) {dbuf[0] = i1;  dbuf[1] = hist[i1];  dbuf[2] = val1;  dbuf[3] = val2;  dbuf[4] = peak; }
    if (i2>=0) {
      float val1 = hist[32+((i2+1)&31)];
      float val2 = hist[32+((i2+31)&31)];
      float peak = i2 + 0.5f + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
      peak = peak * 0.1963495f;
      if (peak <     0) peak += 2*Pi;
      if (peak >= 2*Pi) peak -= 2*Pi;
      d_Orient[bx+maxPts] = peak; // [ 0 ~ 2PI]
//       if (bx==1) {dbuf[8] = i2;  dbuf[9] = hist[i2];  dbuf[10] = val1;  dbuf[11] = val2;  dbuf[12] = peak; }
    } else 
      d_Orient[bx+maxPts] = i2;
  }
} 

double csift_compute_orientations(DevImage *img, DevImage *sift, int numPts, int maxPts, float subsampling)
{
  int w = img->w;
  int h = img->h;
  unsigned int hTimer;
  timer_start(&hTimer);
  float *d_finfo = sift->data;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sift_const), &subsampling, 1*sizeof(float)));
#define OWIDTH  19
#define OHEIGHT 32
//   float h_buf[OWIDTH*OHEIGHT], *dbuf;
//   CUDA_SAFE_CALL(hipMalloc((void **)&dbuf, sizeof(float)*OWIDTH*OHEIGHT));

  dim3 blocks(numPts);
  dim3 threads(OWIDTH);
  th_compute_orientations<<<blocks, threads>>>(img->data, d_finfo, maxPts, w, h);
  CUT_CHECK_ERROR("ComputeOrientations() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  
  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("ComputeOrientations time =    %.2f msec\n", gpuTime);
#endif
//   CUDA_SAFE_CALL(hipMemcpy(h_buf, dbuf, sizeof(float)*OWIDTH*OHEIGHT, hipMemcpyDeviceToHost));
//   hipFree(dbuf);
//   PRINT_ARRAY2(h_buf, OWIDTH, OHEIGHT, "hist", "%5.2f ");
//   PRINT_ARRAY2(h_buf, 8, 4, "hist", "%5.2f ");
  
  return gpuTime;
}

double csift_second_orientations(DevImage *sift, int *initNumPts, int maxPts) 
{
  unsigned int hTimer;
  timer_start(&hTimer);
  int numPts = *initNumPts;
  int numPts2 = 2*numPts;
  float *d_sift = sift->data;
  int bw = sizeof(float)*numPts2;
  float *h_sift = (float *)malloc(7*bw);
  CUDA_SAFE_CALL(hipMemcpy2D(h_sift, bw, d_sift, sizeof(float)*maxPts,  
			      sizeof(float)*numPts, 7, hipMemcpyDeviceToHost));
  int num = numPts;
  for (int i=0;i<numPts;i++) {
    int o2pos = 6 * numPts2 + i;
    if (h_sift[o2pos]>=0.0f && num<maxPts) {
      float *pnew=h_sift+num, *pold=h_sift+i;
      for (int j=0;j<5;j++,pnew+=numPts2,pold+=numPts2) *pnew = *pold;
      h_sift[5*numPts2+num] = h_sift[o2pos];
      h_sift[6*numPts2+num] = -1.0f;
      num ++;
    }
  }
  CUDA_SAFE_CALL(hipMemcpy2D(&d_sift[numPts], sizeof(float)*maxPts, 
			      &h_sift[numPts], bw, sizeof(float)*(num-numPts), 7, hipMemcpyHostToDevice));
  free(h_sift);
  *initNumPts = num;
  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("SecondOrientations time =     %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// Extract Sift descriptors
// ===================================================================

texture<float, 2, hipReadModeElementType> tex;

__global__ void th_extract_descriptors(float *d_finfo, float *d_desc, int maxPts)
{
  const int bx = blockIdx.x;		// bx is just the index of the keypoint
  const int tx = threadIdx.x;		// 16x16 threads for each keypoint
  const int ty = threadIdx.y;		// 16x16 threads for each keypoint
  const int tidx = ty * 16 + tx;
  // get the weighted gradient and orientation for each sample (16x16)
  __shared__ float cont[16*16][8];
//  memset( &cont[tidx][0], 0, 8 * sizeof(float) );
  memset( &cont[tidx][0], 0, 16*16*8 * sizeof(float) );
  {
    float kx = d_finfo[0*maxPts + bx];	// refined x position (on subsampled image)
    float ky = d_finfo[1*maxPts + bx];	// refined y position (on subsampled image)
    float ks = d_finfo[2*maxPts + bx]/sift_const[0];	// refined scale (on subsampled image)
    float aa = d_finfo[5*maxPts + bx];  // orientation in radian
    float radius = ks * 6;		// radius of the coverage of 16 x 16 samples
    float xu[2]={radius/16,0}, yu[2]={0,radius/16};
    float R[4], rot=-Pi/2-aa, rxu[2], ryu[2];
    G2M_SET( R, +cosf(rot), -sinf(rot), +sinf(rot), +cosf(rot) );  // CCW
    G2M_MUL_MV( rxu, R, xu );		// unit vector in X for each sample
    G2M_MUL_MV( ryu, R, yu );		// unit vector in Y for each sample
    float xo = (+tx+ 0.5f - 8);		// [-7.5 +7.5]
    float yo = (-ty+15.5f - 8);		// [-7.5 +7.5]
    float sxy[2];			// position of the sample
    rxu[1] *= -1;  ryu[1] *= -1;
    G2V_SET( sxy, kx, ky );
    G2V_SCALED_ADD( sxy, sxy, 2*xo, rxu );
    G2V_SCALED_ADD( sxy, sxy, 2*yo, ryu );
    float Gx = tex2D(tex, sxy[0]+rxu[0], sxy[1]+rxu[1]) - tex2D(tex, sxy[0]-rxu[0], sxy[1]-rxu[1]);
    float Gy = tex2D(tex, sxy[0]+ryu[0], sxy[1]+ryu[1]) - tex2D(tex, sxy[0]-ryu[0], sxy[1]-ryu[1]);
    float mag = sqrtf(Gx*Gx + Gy*Gy);			// gradient magnitude
    float wgt = exp( - (xo*xo+yo*yo) / (2*8*8) );	// gaussian weight
    float ang = atan2f(Gy,Gx);  if (ang<0) ang += 2*3.1416f;  // gradient orientation [ 0 ~ PI ]
    float anf  = ang * 4.0f / 3.1415926536f;         // orientation bin index [ 0 ~ 8 ]
    int   ani = (int)floor( anf );  if (ani>7) ani -= 8;
    int   an[2];  float anw;
    if (ani+0.5f < anf) { an[0]=ani;   an[1]=ani+1; if (an[1]>7) an[1]=0;  anw=1-fabs(ani+0.5f-anf); }
    else                { an[0]=ani-1; an[1]=ani;   if (an[0]<0) an[0]=7;  anw=  fabs(ani+0.5f-anf); }
    cont[tidx][an[0]] = mag * wgt * anw;
    cont[tidx][an[1]] = mag * wgt * (1-anw);
//     if (bx==0) dbuf[tidx] = Gy;
//     if (bx==0 && tx==0 && ty==0) { 
//       dbuf[0] = kx,     dbuf[1] = ky,     dbuf[2] = ks,     dbuf[3] = aa*180/3.141592f;
//       dbuf[4] = rxu[0]; dbuf[5] = rxu[1]; dbuf[6] = ryu[0]; dbuf[7] = ryu[1];
//       dbuf[8] = sxy[0]; dbuf[9] = sxy[1]; dbuf[16] = xo; dbuf[17] = yo;
//     }
//     if (bx==0 && tx==15 && ty==0)  { dbuf[10] = sxy[0]; dbuf[11] = sxy[1]; dbuf[18] = xo; dbuf[19] = yo; }
//     if (bx==0 && tx== 0 && ty==15) { dbuf[12] = sxy[0]; dbuf[13] = sxy[1]; dbuf[20] = xo; dbuf[21] = yo; }
//     if (bx==0 && tx==15 && ty==15) { dbuf[14] = sxy[0]; dbuf[15] = sxy[1]; dbuf[22] = xo; dbuf[23] = yo; }
//     for (int k=0; k<8; k++) dbuf[tidx] = mag;  ////
//     for (int k=0; k<8; k++) dbuf[tidx*8+k] = (k== an[0] || k==an[1] ? 1 : 0);  // ori
  }
  // initialize the descriptor buffer [ 4 x 4 x 8 ]
  __shared__ float desc[4*4*8];
  if (ty<8) desc[tidx] = 0.0f;
  __syncthreads();
  // add up all contributions from all the samples, for each bin
  if (ty < 8) {
    int bx = tx & 3;	// bx = tx % 4
    int by = tx >> 2;	// by = tx / 4
    int bo = ty;	// bo = ty;
    int bx4=bx*4, by4=by*4, bidx = tx*8+bo, zx, zy, zidx;
//#define ADDS00(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx;                     desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS10(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zx>=0)          desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS01(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zy>=0)          desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS20(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zx<16)          desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS02(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zy<16)          desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS11(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zx>=0 && zy>=0) desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS12(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zx>=0 && zy<16) desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS21(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zx<16 && zy>=0) desc[bidx] += cont[zidx][bo]*ww; } while(0)
//#define ADDS22(xx,yy,ww) do { zx=bx4+xx; zy=by4+yy; zidx=zy*16+zx; if (zx<16 && zy<16) desc[bidx] += cont[zidx][bo]*ww; } while(0)
//    ADDS11(-1,-1, 0.0625f);  ADDS10(-1, 0, 0.1875f);  ADDS10(-1, 1, 0.2500f);
//    ADDS10(-1, 2, 0.2500f);  ADDS10(-1, 3, 0.1875f);  ADDS12(-1, 4, 0.0625f);  // col-1
//    ADDS01( 0,-1, 0.1875f);  ADDS00( 0, 0, 0.5625f);  ADDS00( 0, 1, 0.7500f);
//    ADDS00( 0, 2, 0.7500f);  ADDS00( 0, 3, 0.5625f);  ADDS02( 0, 4, 0.1875f);  // col 0
//    ADDS01( 1,-1, 0.7500f);  ADDS00( 1, 0, 1.0000f);  ADDS00( 1, 1, 1.0000f);
//    ADDS00( 1, 2, 1.0000f);  ADDS00( 1, 3, 1.0000f);  ADDS02( 1, 4, 0.7500f);  // col 1
//    ADDS01( 2,-1, 0.7500f);  ADDS00( 2, 0, 1.0000f);  ADDS00( 2, 1, 1.0000f);
//    ADDS00( 2, 2, 1.0000f);  ADDS00( 2, 3, 1.0000f);  ADDS02( 2, 4, 0.7500f);  // col 2
//    ADDS01( 3,-1, 0.1875f);  ADDS00( 3, 0, 0.5625f);  ADDS00( 3, 1, 0.7500f);
//    ADDS00( 3, 2, 0.7500f);  ADDS00( 3, 3, 0.5625f);  ADDS02( 3, 4, 0.1875f);  // col 3
//    ADDS21( 4,-1, 0.0625f);  ADDS20( 4, 0, 0.1875f);  ADDS20( 4, 1, 0.2500f);
//    ADDS20( 4, 2, 0.2500f);  ADDS20( 4, 3, 0.1875f);  ADDS22( 4, 4, 0.0625f);  // col 4
	///// 
//	zx=bx4-1; zy=by4-1; if (zx>=0 && zy>=0) { zidx=zy*16+zx; desc[bidx] += cont[zidx][bo]*0.0625f; } 			//ADDS11(-1,-1, 0.0625f);
//	zx=bx4-1; if (zx>=0) { zy=by4;   zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS10(-1, 0, 0.1875f);
//	zx=bx4-1; if (zx>=0) { zy=by4+1; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f; }			//ADDS10(-1, 1, 0.2500f);
//	//
//	zx=bx4-1; if (zx>=0) { zy=by4+2; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f; }			//ADDS10(-1, 2, 0.2500f);
//	zx=bx4-1; if (zx>=0) { zy=by4+3; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS10(-1, 3, 0.1875f);
//	zx=bx4-1; zy=by4+4; if (zx>=0 && zy<16) {zidx=zy*16+zx;  desc[bidx] += cont[zidx][bo]*0.0625f; } // col-1	//ADDS12(-1, 4, 0.0625f);
//	////
//	zy=by4-1; if (zy>=0) { zx=bx4;  zidx=zy*16+zx;           desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS01( 0,-1, 0.1875f);
//	zx=bx4; zy=by4; zidx=zy*16+zx;                           desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 0, 0, 0.5625f);
//	zx=bx4; zy=by4+1; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 0, 1, 0.7500f);
//	//
//	zx=bx4; zy=by4+2; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 0, 2, 0.7500f);
//	zx=bx4; zy=by4+3; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 0, 3, 0.5625f);
//	zy=by4+4; if (zy<16) { zx=bx4; zidx=zy*16+zx;            desc[bidx] += cont[zidx][bo]*0.1875f; } // col 0	//ADDS02( 0, 4, 0.1875f);
//	////
//	zy=by4-1; if (zy>=0) { zx=bx4+1; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.7500f; }			//ADDS01( 1,-1, 0.7500f);
//	zx=bx4+1; zy=by4; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 0, 1.0000f);
//	zx=bx4+1; zy=by4+1; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 1, 1.0000f);
//	//
//	zx=bx4+1; zy=by4+2; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 2, 1.0000f);
//	zx=bx4+1; zy=by4+3; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 3, 1.0000f);
//	zy=by4+4; if (zy<16) {zx=bx4+1; zidx=zy*16+zx;           desc[bidx] += cont[zidx][bo]*0.7500f; } // col 1	//ADDS02( 1, 4, 0.7500f);
//	////
//	zy=by4-1; if (zy>=0) {zx=bx4+2; zidx=zy*16+zx;           desc[bidx] += cont[zidx][bo]*0.7500f; }			//ADDS01( 2,-1, 0.7500f);
//	zx=bx4+2; zy=by4; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 0, 1.0000f);
//	zx=bx4+2; zy=by4+1; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 1, 1.0000f);
//	//
//	zx=bx4+2; zy=by4+2; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 2, 1.0000f);
//	zx=bx4+2; zy=by4+3; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 3, 1.0000f);
//	zy=by4+4; if (zy<16) { zx=bx4+2; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.7500f; } // col 2	//ADDS02( 2, 4, 0.7500f);
//	////
//	zy=by4-1; if (zy>=0) { zx=bx4+3; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS01( 3,-1, 0.1875f);
//	zx=bx4+3; zy=by4; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 3, 0, 0.5625f);
//	zx=bx4+3; zy=by4+1; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 3, 1, 0.7500f);
//	//
//	zx=bx4+3; zy=by4+2; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 3, 2, 0.7500f);
//	zx=bx4+3; zy=by4+3; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 3, 3, 0.5625f);
//	zy=by4+4; if (zy<16) { zx=bx4+3; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f; } // col 3	//ADDS02( 3, 4, 0.1875f);  
//	//
//	zx=bx4+4; zy=by4-1; if (zx<16 && zy>=0) {zidx=zy*16+zx;  desc[bidx] += cont[zidx][bo]*0.0625f; }			//ADDS21( 4,-1, 0.0625f);
//	zx=bx4+4; if (zx<16) { zy=by4; zidx=zy*16+zx;            desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS20( 4, 0, 0.1875f);
//	zx=bx4+4; if (zx<16) { zy=by4+1; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f; }			//ADDS20( 4, 1, 0.2500f);
//	//
//	zx=bx4+4; if (zx<16) { zy=by4+2; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f; }			//ADDS20( 4, 2, 0.2500f);	
//	zx=bx4+4; if (zx<16) { zy=by4+3; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS20( 4, 3, 0.1875f);	
//	zx=bx4+4; zy=by4+4; if (zx<16 && zy<16) { zidx=zy*16+zx; desc[bidx] += cont[zidx][bo]*0.0625f; } //col 4    //ADDS22( 4, 4, 0.0625f);
	/////////
	zx=bx4-1; zy=by4-1; if (zx>=0 && zy>=0) { zidx=zy*16+zx; desc[bidx] += cont[zidx][bo]*0.0625f; } 			//ADDS11(-1,-1, 0.0625f);
	          if (zx>=0) { zy=by4;   zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f;				//ADDS10(-1, 0, 0.1875f);
	                       zy=by4+2; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f;				//ADDS10(-1, 2, 0.2500f);
	                       zy=by4+1; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f;				//ADDS10(-1, 1, 0.2500f);
	                       zy=by4+3; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS10(-1, 3, 0.1875f);
	          zy=by4+4; if (zx>=0 && zy<16) {zidx=zy*16+zx;  desc[bidx] += cont[zidx][bo]*0.0625f; } // col-1	//ADDS12(-1, 4, 0.0625f);
	////
	zx=bx4; zy=by4; zidx=zy*16+zx;                           desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 0, 0, 0.5625f);
	zy=by4-1; if (zy>=0) { zidx=zy*16+zx;                    desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS01( 0,-1, 0.1875f);
	        zy=by4+1; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 0, 1, 0.7500f);
	//
	        zy=by4+2; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 0, 2, 0.7500f);
	        zy=by4+3; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 0, 3, 0.5625f);
	zy=by4+4; if (zy<16) { zidx=zy*16+zx;                    desc[bidx] += cont[zidx][bo]*0.1875f; } // col 0	//ADDS02( 0, 4, 0.1875f);
	////
	zx=bx4+1; zy=by4; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 0, 1.0000f);
	zy=by4-1; if (zy>=0) { zidx=zy*16+zx;                    desc[bidx] += cont[zidx][bo]*0.7500f; }			//ADDS01( 1,-1, 0.7500f);
	          zy=by4+1; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 1, 1.0000f);
	//
	          zy=by4+2; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 2, 1.0000f);
	          zy=by4+3; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 1, 3, 1.0000f);
	zy=by4+4; if (zy<16) {zidx=zy*16+zx;                     desc[bidx] += cont[zidx][bo]*0.7500f; } // col 1	//ADDS02( 1, 4, 0.7500f);
	////
	zx=bx4+2; zy=by4; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 0, 1.0000f);
	zy=by4-1; if (zy>=0) {zidx=zy*16+zx;                     desc[bidx] += cont[zidx][bo]*0.7500f; }			//ADDS01( 2,-1, 0.7500f);
	          zy=by4+1; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 1, 1.0000f);
	//
	          zy=by4+2; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 2, 1.0000f);
	          zy=by4+3; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo];						//ADDS00( 2, 3, 1.0000f);
	zy=by4+4; if (zy<16) { zidx=zy*16+zx;                    desc[bidx] += cont[zidx][bo]*0.7500f; } // col 2	//ADDS02( 2, 4, 0.7500f);
	////
	zx=bx4+3; zy=by4; zidx=zy*16+zx;                         desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 3, 0, 0.5625f);
	zy=by4-1; if (zy>=0) { zidx=zy*16+zx;                    desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS01( 3,-1, 0.1875f);
	          zy=by4+1; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 3, 1, 0.7500f);
	//
	          zy=by4+2; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo]*0.7500f;				//ADDS00( 3, 2, 0.7500f);
	          zy=by4+3; zidx=zy*16+zx;                       desc[bidx] += cont[zidx][bo]*0.5625f;				//ADDS00( 3, 3, 0.5625f);
	zy=by4+4; if (zy<16) { zidx=zy*16+zx;                    desc[bidx] += cont[zidx][bo]*0.1875f; } // col 3	//ADDS02( 3, 4, 0.1875f);  
	//
	zx=bx4+4; zy=by4-1; if (zx<16 && zy>=0) {zidx=zy*16+zx;  desc[bidx] += cont[zidx][bo]*0.0625f; }			//ADDS21( 4,-1, 0.0625f);
	          if (zx<16) { zy=by4; zidx=zy*16+zx;            desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS20( 4, 0, 0.1875f);
	          if (zx<16) { zy=by4+1; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f; }			//ADDS20( 4, 1, 0.2500f);
	//
	          if (zx<16) { zy=by4+2; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.2500f; }			//ADDS20( 4, 2, 0.2500f);	
	          if (zx<16) { zy=by4+3; zidx=zy*16+zx;          desc[bidx] += cont[zidx][bo]*0.1875f; }			//ADDS20( 4, 3, 0.1875f);	
	          zy=by4+4; if (zx<16 && zy<16) { zidx=zy*16+zx; desc[bidx] += cont[zidx][bo]*0.0625f; } //col 4    //ADDS22( 4, 4, 0.0625f);
  }
  __syncthreads();
  
  if (true) {	// normalize the description vector  ////
    __shared__ float sum[4*16];  // 4*16 == 4*4*8/2
    // calculate the sum for normalization
    if (ty < 4) {
    	sum[tidx] = desc[tidx] + desc[tidx + 4*16];
	}
	__syncthreads();
    if (ty < 2) {
    	sum[tidx] += sum[tidx + 2*16];
	}
	__syncthreads();
    if (ty < 1) sum[0] = ( sum[0] + sum[1] + sum[2] + sum[3] + sum[4] + sum[5] + 
			   sum[6] + sum[7] + sum[8] + sum[9] + sum[10] + sum[11] + 
			   sum[12] + sum[13] + sum[14] + sum[15] );
    __syncthreads();
    // normalize the descriptor
    if (ty < 8) { 
      float val = desc[tidx] / sum[0]; 		// normalize the descriptor
      desc[tidx] = (val > 0.2f ? 0.2f : val);	// cut off the values bigger than 0.2;
    }
    // calculate the sum for normalization
    if (ty < 4) sum[tidx] = desc[tidx] + desc[tidx + 4*16];
    __syncthreads();
    if (ty < 2) sum[tidx] += sum[tidx + 2*16];
    __syncthreads();
    if (ty==0 && tx==0) sum[0] = ( sum[0] + sum[1] + sum[2] + sum[3] + sum[4] + sum[5] + 
				   sum[6] + sum[7] + sum[8] + sum[9] + sum[10] + sum[11] + 
				   sum[12] + sum[13] + sum[14] + sum[15] );
    __syncthreads();
    // normalize the descriptor
    if (ty < 8) {
    	desc[tidx] /= sum[0]; 		// normalize the descriptor
   	}
  }
  
  // copy the descriptor
  if (ty < 8) {
    float *g_desc = d_desc + 128 * bx;
    g_desc[tidx] = desc[tidx];
  }
}

void save_fbuf(float *data, int w, int h, char *fname);

double csift_extract_descriptors(DevTexture *timg, DevImage *sift,
				 DevImage *desc, int numPts, int maxPts, float subsampling)
{
  unsigned int hTimer;
  timer_start(&hTimer);
  float *d_sift = sift->data, *d_desc = desc->data;
  
  tex.addressMode[0] = hipAddressModeClamp;
  tex.addressMode[1] = hipAddressModeClamp;
  tex.filterMode = hipFilterModeLinear; 
  tex.normalized = false;
  float values[]={ subsampling, timg->w, timg->h };
  CUDA_SAFE_CALL(hipBindTextureToArray(tex, (hipArray*)timg->array));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sift_const), values, 3*sizeof(float)));
  
  dim3 blocks(numPts); 
  dim3 threads(16, 16);
//   float *dbuf=NULL;							// for debugging
//   CUDA_SAFE_CALL(hipMalloc((void**)&dbuf, 16*16*8*sizeof(float)));	// for debugging

//printf("maxPts = %d\n", maxPts);
  th_extract_descriptors<<<blocks, threads>>>(d_sift, d_desc, maxPts);
  
//   // test texture
//   int w = timg->w, h = timg->h;
//   float *d_img=NULL;  hipMalloc((void**)&d_img, w*h*sizeof(float));
//   dim3 blk(iDivUp(w, 16), iDivUp(h,16)), thr(16,16);
//   TestTexture <<< blk, thr >>> (d_img, w, h, 3.14f/4);
//   float *h_img = (float*)malloc(w*h*sizeof(float));
//   hipMemcpy( h_img, d_img, w*h*sizeof(float), hipMemcpyDeviceToHost);
//   save_fbuf( h_img, w, h, "output.png" );
 
  CUT_CHECK_ERROR("ExtractSiftDescriptors() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipUnbindTexture(tex));

  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("ExtractSiftDescriptors time = %.2f msec\n", gpuTime);
#endif
//   float hbuf[16*16*8], hdesc[128];
//   CUDA_SAFE_CALL(hipMemcpy(hbuf,  dbuf, 16*16*8*sizeof(float), hipMemcpyDeviceToHost));
//   CUDA_SAFE_CALL(hipMemcpy(hdesc, d_desc, 128*sizeof(float), hipMemcpyDeviceToHost));
//   CUDA_SAFE_CALL(hipFree(dbuf));
//   PRINT_ARRAY2(hbuf, 8, 3, "dbuf", "%7.4f ");
//   PRINT_ARRAY2(hbuf, 16, 16, "dbuf", "%7.4f ");
//   PRINT_ARRAY3(hbuf, 16, 16, 8, "dbuf", "%.0f");
//   PRINT_ARRAY2(hdesc, 8, 16, "desc", "%7.4f ");
  
  return gpuTime; 
}

// ===================================================================
// ===================================================================

double csift_add_sift_data(HstImage *flist, float *d_sift, float *d_desc,
			   int numPts, int maxPts, float subsampling)
{
  unsigned int hTimer;
  timer_start(&hTimer);
  // scale back XY position of the features (Not the scale)
  float *buffer = (float *)malloc(sizeof(float)*numPts*6);
  int bwidth = sizeof(float)*numPts;
  CUDA_SAFE_CALL(hipMemcpy2D(buffer, bwidth, d_sift, sizeof(float)*maxPts, bwidth, 6, hipMemcpyDeviceToHost));
  for (int i=0;i<2*numPts;i++) buffer[i] *= subsampling;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  // increase the size of the host/device memories, if necessary
  int newNum = flist->h + numPts;
  if (newNum > flist->hmax) flist->reallocImage( 140, newNum*2 );
  // copy all the features
  //   HstImage 'flist' is 140 x N float image, with a SIFT feature in each row.
  //   Each SIFT feature consists of 140 float values, including 128 descriptor.
  //   ( x,y,s,sh,  edge,ori,sco,amb,  desc[128], dummy[4] )
  float *ptr   = flist->data + flist->h * 140;
  // copy each row of 'd_sift' to a column of 'flist'
  for (int j=0; j<numPts; j++) {	// each keypoint
    float *fp = ptr + j * 140;
    for (int i=0; i<6; i++) fp[i] = buffer[ i * numPts + j ];
    fp[6] = fp[7] = 0.0;
    // note that orientation is in radian !
  }
  // copy each row of 'd_desc' to row of 'flist'
  CUDA_SAFE_CALL(hipMemcpy2D(&ptr[8], 140*sizeof(float),
			      d_desc, 128*sizeof(float), 128*sizeof(float), numPts, hipMemcpyDeviceToHost));
  flist->setImage( 140, newNum );
  free(buffer);
  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("AddSiftData time =            %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// ===================================================================

void save_fbuf(float *data, int w, int h, char *fname);

void check_img(DevImage *cimg, char *cmmt, int oct, int lvl, bool print_on, bool save_on)
{
  int   i, w=cimg->w, h=cimg->h;
  int   total = w*h, cnt_zero=0, cnt_nega=0, cnt_posi=0;
  HstImage himg;
  himg.setImage( cimg->w, cimg->h );
  himg.uploadFrom( cimg );
  float *imd = himg.data;
  float maxv=-1e6, minv=+1e6;
  for (i = 0; i < total; i++) {
    if (imd[i] < minv) minv = imd[i];
    if (imd[i] > maxv) maxv = imd[i];
    if      (imd[i] == 0) cnt_zero++;
    else if (imd[i] > +0.0001) cnt_posi++;
    else if (imd[i] < -0.0001) cnt_nega++;
  }
  if (print_on) printf("%s (%3d x %3d)  minv=%7.4f, maxv=%7.4f  (zero:%4.1f%%  pos:%4.1f%%  neg:%4.1f%%)\n",
		       (cmmt ? cmmt:"BUF"), w, h, minv, maxv, cnt_zero*100.0f/total, cnt_posi*100.0f/total, cnt_nega*100.0f/total);
  if (save_on) {
    char fname[80];  sprintf( fname, "%s_o%d_l%d.png", cmmt, oct, lvl );
    save_fbuf( imd, w, h, fname );
  }
}

void check_sift(DevImage *sift, int cnt, int *ptrs, int iw, int ih)
{
  int   i, w=sift->w;
  HstImage himg;
  himg.setImage( sift->w, sift->h );
  himg.uploadFrom( sift );
  float *imd = himg.data;
  printf("  keys = %d   width=%d   image=(%dx%d)\n", cnt, w, iw, ih);
  for (i = 0; i < cnt; i++) {
    if (!ptrs) {
      printf("  key xy=(%.2f %.2f) s=%.2f val=%.4f score=%.4f o=%.0f\n", imd[i+0*w], imd[i+1*w], imd[i+2*w], imd[i+3*w], imd[i+4*w], imd[i+5*w]*180/3.141592f);
    } else {
      printf("  key xy=(%.2f %.2f) s=%.2f val=%.4f score=%.4f o=%.0f  ptrs[i]=%d(%d,%d)\n", imd[i+0*w], imd[i+1*w], imd[i+2*w], imd[i+3*w], imd[i+4*w], imd[i+5*w]*180/3.141592f, ptrs[i], ptrs[i]%iw, ptrs[i]/iw);
    }
  }
}


