#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cutil.h>
#include "cuda_image_edit.h"

__device__ __constant__ float edit_const[2]; 

// ===================================================================
// 
// ===================================================================

// __global__ void th_sample_up(float *dst, float *src, int sw, int sh)
// {
//   // Upsample the 'src' and set the pixels of 'dst', which is two times bigger.
//   //   'sw' and 'sh' is the size of 'src', not 'dst', and 
//   //   the thread blocks(16x16) must divide up the size of 'dst'.
//   const int dw = __mul24(2,sw), dh = __mul24(2,sh);
//   const int dx = __mul24(16, blockIdx.x) + threadIdx.x;
//   const int dy = __mul24(16, blockIdx.y) + threadIdx.y;
//   if (dx >= dw || dy >= dh) return;
//   int   sx = dx>>1, sy = dy>>1;
//   int spos = __mul24(sy, sw) + sx;
//   dst[ __mul24(dy, dw) + dx ] = src[spos];
//   __syncthreads();
// }

__global__ void th_sample_up(float *dst, float *src, int dw, int dh)
{
  // Upsample the 'src' and set the pixels of 'dst', which is two times bigger.
  //   'dw' and 'dh' is the size of 'dst', not 'src', and 
  //   the thread blocks(16x16) must divide up the size of 'dst'.
//   if (blockIdx.x != 1 || blockIdx.y != 1) return;
  __shared__ float data[9][9];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int dx = __mul24(16, blockIdx.x) + threadIdx.x;
  int dy = __mul24(16, blockIdx.y) + threadIdx.y;
  int sstt = __mul24(dy>>1, dw>>1) + __mul24(8, blockIdx.x);
  int tx2 = tx>>1;
  int ty2 = ty>>1;
  int didx = __mul24(dy, dw) + dx;
  if (dx >= 200 || dy >= 200) return;
  if (tx < 9 && ty < 9 && dx < dw && dy < dh) data[ty][tx] = src[sstt + tx];
#if 1		//// Why is it failing ?
  if (tx&0x1 == 0) {
    if (ty&0x1 == 0) dst[didx] = (data[ty2][tx2]);
    else             dst[didx] = (data[ty2][tx2] + data[ty2+1][tx2]) / 2;
  } else {
    if (ty&0x1 == 0) dst[didx] = (data[ty2][tx2] + data[ty2][tx2+1]) / 2;
    else             dst[didx] = (data[ty2][tx2] + data[ty2][tx2+1] + data[ty2+1][tx2] + data[ty2+1][tx2+1]) / 4;
  }
#else		//// ... while this code is OK ?
  dst[__mul24(dy, dw) + dx] = src[ (dy>>1)*(dw>>1) + (dx>>1) ];
#endif
  __syncthreads();
}

double cimg_scale_up(DevImage *res, DevImage *src)
{
  int w = src->w;
  int h = src->h;
  if (res->data==NULL || src->data==NULL) {
    // printf("cimg_scale_up: missing data\n");
    return 0.0;
  }
  unsigned int hTimer; 
  timer_start(&hTimer);
  int  dw = w*2,  dh = h*2;
#if 0	//// This code is not working at all. Don't know why...
  dim3 blocks(iDivUp(dw,16), iDivUp(dh,16));
  dim3 threads(16,16);
  th_sample_up<<<blocks, threads>>>(res->data, src->data, dw, dh); 
  CUT_CHECK_ERROR("cimg_scale_up() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());
#else
  HstImage hsrc(w, h+1);
  HstImage hres(dw, dh);
  src->uploadTo( hsrc.data );
  int   x, y, spos, dpos, dnxt;
  // duplicate the last row
  float *py = hsrc.data + (h-1)*w, *pz = hsrc.data + h*w;
  for (x = 0; x < w; x++) pz[x] = py[x];  
  // 
  for (y = 0; y < h-1; y++) {
    for (x = 0; x < w; x++) {
      spos =   y *  w +   x;
      dpos = 2*y * dw + 2*x;
      dnxt = (2*y+1) * dw + 2*x;
      hres.data[dpos]   = (hsrc.data[spos]);
      hres.data[dpos+1] = (hsrc.data[spos] + hsrc.data[spos+1])/2;
      hres.data[dnxt]   = (hsrc.data[spos] + hsrc.data[spos+w])/2;
      hres.data[dnxt+1] = (hsrc.data[spos] + hsrc.data[spos+1] + hsrc.data[spos+w] + hsrc.data[spos+w+1])/4;
    }
  }
  // duplicate the last column
  pz = hres.data + dw-1;
  for (y = 0; y < dh-1; y++, pz+=dw) pz[0] = pz[-1];
  // 
  res->downloadFrom( hres.data );
#endif

  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("cimg_scale_up time =              %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// 
// ===================================================================

__global__ void th_sample_down(float *dst, float *src, int width, int height)
{
  // Downsample the 'src' and set the pixels of 'dst', which is half of the size.
  //   'width' and 'height' is the size of 'src', not 'dst', and 
  //   the thread blocks(16x16) must divide up the size of 'dst'.
  const int hw = width/2, hh = height/2;
  const int x = __mul24(16, blockIdx.x) + threadIdx.x;
  const int y = __mul24(16, blockIdx.y) + threadIdx.y;
  if (x >= hw || y >= hh) return;
  int dpos = __mul24(2, __mul24(y, width)) + __mul24(2, x);
  float sum = (src[dpos] + src[dpos+1] + src[dpos+width] + src[dpos+width+1])/4;
  dst[ __mul24(y, hw) + x ] = sum;
  __syncthreads();
}

double cimg_scale_down(DevImage *res, DevImage *src, float variance)
{
  int w = src->w;
  int h = src->h;
  if (res->data==NULL || src->data==NULL) {
    // printf("cimg_scale_down: missing data\n");
    return 0.0;
  }
  unsigned int hTimer; 
  timer_start(&hTimer);
  int  hw = w/2,  hh = h/2;
  dim3 blocks(iDivUp(hw,16), iDivUp(hh,16));
  dim3 threads(16,16);
  th_sample_down<<<blocks, threads>>>(res->data, src->data, w, h); 
  CUT_CHECK_ERROR("cimg_scale_down() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("cimg_scale_down time =              %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// 
// ===================================================================

__global__ void th_copy(float *d_dst, float *d_src, int width, int height)
{
  const int x = __mul24(blockIdx.x, 16) + threadIdx.x;
  const int y = __mul24(blockIdx.y, 16) + threadIdx.y;
  int p = __mul24(y, width) + x;
  if (x<width && y<height) d_dst[p] = d_src[p];
  __syncthreads();
}

double cimg_copy(DevImage *dst, DevImage *src)
{
  int w = dst->w;
  int h = dst->h;
  if (dst->data==NULL || src->data==NULL) {
    // printf("cimg_copy: missing data\n");
    return 0.0;
  }
  unsigned int hTimer;
  timer_start(&hTimer);
  dim3 blocks(iDivUp(w, 16), iDivUp(h, 16));
  dim3 threads(16, 16);
  th_copy<<<blocks, threads>>>(dst->data, src->data, w, h);
  CUT_CHECK_ERROR("cimg_copy() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("cimg_copy time =               %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// 
// ===================================================================

__global__ void th_subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
  const int x = __mul24(blockIdx.x, 16) + threadIdx.x;
  const int y = __mul24(blockIdx.y, 16) + threadIdx.y;
  int p = __mul24(y, width) + x;
  if (x<width && y<height) d_Result[p] = d_Data1[p] - d_Data2[p];
  __syncthreads();
}

double cimg_subtract(DevImage *res, DevImage *dataA, DevImage *dataB)
{    
  int w = res->w;
  int h = res->h;
  unsigned int hTimer;
  timer_start(&hTimer);
  if (res->data==NULL || dataA->data==NULL || dataB->data==NULL) {
    // printf("cimg_subtract: missing data\n");
    return 0.0;
  }
  dim3 blocks(iDivUp(w, 16), iDivUp(h, 16));
  dim3 threads(16, 16);
  th_subtract<<<blocks, threads>>>(res->data, dataA->data, dataB->data, 
				w, h);
  CUT_CHECK_ERROR("cimg_subtract() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("cimg_subtract time =               %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}

// ===================================================================
// 
// ===================================================================

__global__ void th_multiply_add(float *d_Result, float *d_Data, int width, int height)
{
  const int x = __mul24(blockIdx.x, 16) + threadIdx.x;
  const int y = __mul24(blockIdx.y, 16) + threadIdx.y;
  int p = __mul24(y, width) + x;
  if (x<width && y<height) d_Result[p] = edit_const[0]*d_Data[p] + edit_const[1];
  __syncthreads();
}

double cimg_multiply_add(DevImage *res, DevImage *data, float constA, float constB)
{
  int w = res->w;
  int h = res->h;
  if (res->data==NULL || data->data==NULL) {
    // printf("cimg_multiply_add: missing data\n");
    return 0.0;
  }
  unsigned int hTimer;
  float values[2] = { constA, constB };
  timer_start(&hTimer);
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edit_const), &values, 2*sizeof(float)));

  dim3 blocks(iDivUp(w, 16), iDivUp(h, 16));
  dim3 threads(16, 16);
  th_multiply_add<<<blocks, threads>>>(res->data, data->data, w, h);
  CUT_CHECK_ERROR("cimg_multiply_add() execution failed\n");
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  double gpuTime = timer_stop(hTimer);
#ifdef VERBOSE
  printf("cimg_multiply_add time =            %.2f msec\n", gpuTime);
#endif
  return gpuTime;
}
